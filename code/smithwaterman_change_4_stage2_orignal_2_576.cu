#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <unistd.h>
#include <omp.h>
#include<limits>
 double diff(timespec start, timespec end)
 {
 	double a=0;
        if((end.tv_nsec-start.tv_nsec)<0)
        {
        a=end.tv_sec-start.tv_sec-1;
        a+=(1000000000+end.tv_nsec-start.tv_nsec)/1000000000.0;
        }
        else
        {
        a=end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

        }
        return a;

}

struct NUM_ADD
{
                short2 read_reference_number;
                int address_array;
};


  __global__ void  calculate_cigar( int size, char * data, NUM_ADD *num_add,int4 * result, int * direction) //, char * result
{
	int offset=blockIdx.x;
	__shared__ short2 read_reference_number;
	__shared__ char * read_base_array;
	__shared__ char4 * reference_base_array;
	__shared__ int mismatch;
	__shared__ int  match;
	__shared__ int open;
	__shared__ int extend;
	__shared__ short2 * direction_index;
	while(offset<size)
         {
		if( threadIdx.x==0)
                {
                read_reference_number=num_add[offset].read_reference_number;
                read_base_array=(char *) (data+num_add[offset].address_array);
	        reference_base_array=(char4 *) (read_base_array+(read_reference_number.x+127)/128*128);
		direction_index=(short2 *) (direction+offset*640*1100);  
	     	}
                __syncthreads();

 		__shared__ char reference_base_in_char[600];
                int hh=(read_reference_number.y+4-1)/4;
                int tt=(hh+blockDim.x-1)/blockDim.x;
                for(int ii=0;ii<tt;ii++)
                {
                        int aa=threadIdx.x+ii*blockDim.x;
                        if(aa< hh)
                        {
                        char4 reference_base_in_thread;
                        reference_base_in_thread=reference_base_array[aa]; //Is it right to get data from global memory
                        reference_base_in_char[aa*4]=reference_base_in_thread.x;
                        reference_base_in_char[aa*4+1]=reference_base_in_thread.y;
                        reference_base_in_char[aa*4+2]=reference_base_in_thread.z;
                        reference_base_in_char[aa*4+3]=reference_base_in_thread.w;
			 }
                }
		
		__shared__ int MM[578]; 
                __shared__ int gap_h[578]; //insertion
                __shared__ short2 gap_size_h[578];  //insertion
	 	__shared__ int result_col;
		__shared__ int result_row;
		__shared__ int result_col_index;
		__shared__ int result_row_index;
		//__shared__ char cigar_m[128];
		//__shared__ int cigar_int_m[128];
		//int final_result;
	        //int final_i;
	        //int final_j;


                if(threadIdx.x==0)
                {
                        MM[0]=0;
                        gap_h[0]=-1000000000;//std::numeric_limits<int>::min()/2;
                        gap_size_h[0].x=0;
			gap_size_h[0].y=0;
			match=200;
			mismatch=-150;
			open=-260;
			extend=-11;
			result_col=-1000000000;//std::numeric_limits<int>::min()/2;
			result_row=-1000000000;//std::numeric_limits<int>::min()/2;
                //	for(int i=0;i<read_reference_number.y;i++)
		//	printf("%c",reference_base_in_char[i]);
		//	printf("\n");
		//	for(int i=0;i<read_reference_number.x;i++)
		//	printf("%c",read_base_array[i]);
	//printf("%d\n",offset);
		}
		 __syncthreads();

	//	int read_number=read_reference_number.x;
                {
			
			char read_base;
                        read_base=read_base_array[threadIdx.x];

			int gap_v=-1000000000;//std::numeric_limits<int>::min()/2;;
			int gap_size_v=0; //Deletion
 			int M=0; //now 
                        int step_right; //now
			int ki=0;//insertion  h  negetive
			 //deletion  v
			int MMM=0; 
                        short mt=0; 
			short2 curmt;
			curmt.x=0;
			curmt.y=0;               
			int current_reference_id=0;
                        for(int j=0;j<read_reference_number.x+read_reference_number.y-1;j++)
                        {
                                int aa=j-threadIdx.x;
                                if( aa>=0 && (current_reference_id<read_reference_number.y))
                                {
					int prev_gap=M+open; //M which is cacluated by last step in the same thread
					gap_v+=extend;
					if(prev_gap>gap_v)
					{
						gap_v=prev_gap;
						gap_size_v=1;
					}	
					else
						gap_size_v++;
					
					char reference_base_each=reference_base_in_char[current_reference_id];
				
					M=MMM+(read_base==reference_base_each? match:mismatch);
					prev_gap=MM[threadIdx.x]+open;
					step_right=gap_h[threadIdx.x]+extend;
					if(prev_gap>step_right)
					{
						step_right=prev_gap;
						ki=1;
					}	
					else
						ki=gap_size_h[threadIdx.x].x+1;

					bool diag=(M>=gap_v)&&(M>=step_right);
					curmt.y=0;
				
					if(diag)
					{
					curmt.x=0;
					//if(threadIdx.x==0||current_reference_id==0)
					//	curmt.y=0;
				//	else
						curmt.y=mt+1;
					//  curBtrack=0;
					}
					else
					if(step_right>=gap_v)
					{
						 M=step_right;
						curmt.x=0-ki;
						// curBtrack=0-ki;
					}
					else
						{
							M=gap_v;
							curmt.x=gap_size_v;
							//curBtrack=gap_size_v;
						}
					MMM=MM[threadIdx.x];
					mt=gap_size_h[threadIdx.x].y;
					direction_index[640*j+threadIdx.x]=curmt;
			//if(threadIdx.x==read_reference_number.x-3)
				//printf("%p %d ", &direction_index[800*j+threadIdx.x],curBtrack);
 				
				if(current_reference_id==read_reference_number.y-1)
				{	
					if(M>=result_row)
					{
						result_row=M;
						result_row_index=threadIdx.x;  //
					}
					//printf("%d %d  %d  %d %d \n",read_reference_number.y,M,result_row,result_row_index,threadIdx.x);
				}
                         	if(threadIdx.x==read_reference_number.x-1)
                                {
						if(M>=result_col)
						{
							result_col=M;
							result_col_index=current_reference_id;	// +1					
						}						
				}

				current_reference_id++;
		

			       }
                        	
				__syncthreads(); //to make sure that the former value of MM[threadIdx.x+1] are used by other threads.
                                MM[threadIdx.x+1]=M;
                                gap_h[threadIdx.x+1]=step_right;
                                gap_size_h[threadIdx.x+1].x=ki;
				gap_size_h[threadIdx.x+1].y=curmt.y;
                                __syncthreads(); // there should be two synthreads(); // to make sure that all of MM[threadIdx.x+1] have get a new value before M,D and I changed.
                        }
                }
	//	char state;//0  match;  1 mistmatch; 2 inseriton;  3  deletion
	//	__shared__ int cigar_index;
	//	int segment_length;
	//	short2 btr;
          //     	char  new_state;
	//	int step_length;
		//if(threadIdx.x==0)printf("Offset=%d\n",offset);
		int4 result4;
	//	 if(threadIdx.x==read_reference_number.x-1)
        	if(threadIdx.x==0)  
	      {
			//printf("%d %d %d %d\n", result_row,result_col, result_row_index,result_col_index);
                        if(result_row>result_col||result_row==result_col&&(read_reference_number.x-result_row_index-1)>(read_reference_number.y-result_col_index-1))
			{
			//	final_result=result_row;			
				result4.x=read_reference_number.y-1;
				result4.y=result_row_index;
				result4.z=read_reference_number.x-1-result_row_index;
			}
			else
			{
			//	final_result=result_col;
				result4.x=result_col_index;
				result4.y=read_reference_number.x-1;
				result4.z=0;
			}
			//result[offset*3]=final_result;
	//	printf("%d %d %d %d\n",size,offset,result4.x,result4.y);
			result[offset]=result4;
		}
                __syncthreads();
		offset+=gridDim.x;
		//if(threadIdx.x==0)		printf("%d %d\n",offset,size);
	}
}




  __global__ void  calculate_cigar_2( int size, int4 * result, char * cigar,int * cigar_int,int * direction) //, char * result
{
	int offset=blockIdx.x;
	int4 result4;;
	 short2 * direction_index;
	__shared__ char * cigar_store;
	__shared__ int *cigar_int_store;
	__shared__ char cigar_m[128];
        __shared__ int cigar_int_m[128];
	while(offset<size)
         {
		 char state;//0  match;  1 mistmatch; 2 inseriton;  3  deletion
                __shared__ int cigar_index;
                int segment_length;
                short2 btr;
                char  new_state;
    	        int step_length;

		if( threadIdx.x==0)
                {
                	result4=result[offset];
			direction_index=(short2 *) (direction+offset*640*1100);  
			cigar_store=(char *) (cigar+offset*sizeof(char)*128);
			cigar_int_store=(int *) (cigar_int+offset*128);

			//printf("\n %d %d\n", final_i,final_j);
 
               		cigar_index=0;	
			if(result4.z>0)
			{
			cigar_m[cigar_index]='S';
			cigar_int_m[cigar_index]=result4.z;
			cigar_index++;
			}
			segment_length=0;
			state='N';
			do
			{
				btr=direction_index[(result4.x+result4.y)*640+result4.y];
				if(btr.x>0)
				{
					new_state='D';
					step_length=btr.x;
					result4.x-=step_length;
				}
				else
				if(btr.x<0)
				{
					new_state='I';
					step_length=0-btr.x;		
					result4.y-=step_length;
				}
				else	
				{
					new_state='M';
					step_length=btr.y;
					result4.x-=step_length;
					result4.y-=step_length;
			
				}
				
				if(state=='N') state=new_state;
				if(state==new_state) 
				{
					segment_length+=step_length;
				}
				else
				{
				 cigar_m[cigar_index]=state;
                       		 cigar_int_m[cigar_index]=segment_length;
                       		 segment_length=step_length;
                       		 cigar_index++;
			 	 state=new_state;
				}
	
			}while(result4.x>=0&&result4.y>=0);
                       	
			cigar_m[cigar_index]=state;
                       	cigar_int_m[cigar_index]=segment_length;
                       	cigar_index++;
			if(result4.y>=0) 
			{	
				cigar_m[cigar_index]='S';
				cigar_int_m[cigar_index]=result4.y+1;
				cigar_index++;
			}	

			result4.z=result4.x+1;
			result4.w=cigar_index;
			result[offset]=result4;
	/*		for(int i=cigar_index-1;i>=0;i--)
			{
			printf("%d%c",cigar_int_m[i],cigar_m[i]);
			}
*/
		 }
		 __syncthreads();
	
		if(threadIdx.x<cigar_index && cigar_index<=blockDim.x)
		{
	//	if(threadIdx.x==0)
	//		printf("%c %d\n",cigar_m[cigar_index-1-threadIdx.x], cigar_int_m[cigar_index-1-threadIdx.x]);
		cigar_store[threadIdx.x]=cigar_m[cigar_index-1-threadIdx.x];
		cigar_int_store[threadIdx.x]=cigar_int_m[cigar_index-1-threadIdx.x];
	//	if(threadIdx.x==0)
	//		printf("%c %d\n", cigar_store[threadIdx.x],cigar_int_store[threadIdx.x]);
		
		}

		offset+=gridDim.x;
	}
}



struct InputData
{
char read_base[600];
char reference_base[600];
};


int main(int artc, char* args[])
{
	int total_size=0;
	FILE * file;
        file=fopen(args[1],"r");
        int size;
	double  computation_time=0;//total_time=0;
	timespec start,finish;	 
     /*  char data[200][1000];
                for(int i=0;i<101;i++)
                {
                        fscanf(file,"%s ", data[i]);
                }
                int row=atoi(args[2]);
                int col=atoi(args[3]);
                size=row*col;
        for(int ww=0;ww<1;ww++)
        {       int index=0;
                InputData * inputdata=(InputData* )malloc(size*(sizeof(InputData)));
                for(int i=0;i<row;i++)
                for(int j=0;j<col;j++)
                {
                        strcpy(inputdata[index].reference_base,data[1]);
                        strcpy(inputdata[index].read_base,data[1]);
                        index++;
                }       

       */

 /*       fscanf(file,"%d",&size);
	 while(!feof(file))
        {
                InputData * inputdata=(InputData* )malloc(size*(sizeof(InputData)));
                for(int i=0;i<size;i++)
                {
                        fscanf(file,"%s ",inputdata[i].reference_base);
                        fscanf(file,"%s ",inputdata[i].read_base);
                }
   */
	 char data[200][1000];
                for(int i=0;i<101;i++)
                {
                        fscanf(file,"%s ", data[i]);
                }
                int row=atoi(args[2]);
                int col=atoi(args[3]);
                size=row*col;
        for(int ww=0;ww<1;ww++)
        {       int index=0;
                InputData * inputdata=(InputData* )malloc(size*(sizeof(InputData)));
                for(int i=0;i<row;i++)
                for(int j=0;j<col;j++)
                {
                        strcpy(inputdata[index].reference_base,data[i]);
                        strcpy(inputdata[index].read_base,data[j]);
                        index++;
                }
     
		//data preparation.
		char * data_h_total=(char*)malloc(size * 640* sizeof (char)*2+(size*sizeof(NUM_ADD)+127)/128*128);
		NUM_ADD * data_num_add=(NUM_ADD *) (data_h_total);
		char * data_h=data_h_total+(size*sizeof(NUM_ADD)+127)/128*128;  //.thus we donot need to worry about align
		int data_size=0;
		char * data_d_total;		
		hipMalloc( (char **) &data_d_total, (size*sizeof(NUM_ADD)+127)/128*128+size *( 640 )* sizeof (char)*2+sizeof(int)*size*4);
		int * result_h=(int *) malloc(sizeof(int)*size*4);
	 	char * cigar_h=(char *) malloc(sizeof(char)*size*128);
		int * cigar_int_h=(int *) malloc(sizeof(int)*size*128);
		for(int i=0;i<size;i++)
		{

			char4 reference_tep[150];
			int read_len=strlen(inputdata[i].read_base);
			int ref_len=strlen(inputdata[i].reference_base);
			int new_len=(ref_len+4-1)/4;
			total_size+=ref_len*read_len;
			for(int j=0;j<new_len;j++)
		        {
		        	reference_tep[j].x=inputdata[i].reference_base[j*4];
		                if(j*4+1<ref_len)
		                reference_tep[j].y=inputdata[i].reference_base[j*4+1];
		                if(j*4+2<ref_len)
		                reference_tep[j].z=inputdata[i].reference_base[j*4+2];
		                if(j*4+3<ref_len)
		                reference_tep[j].w=inputdata[i].reference_base[j*4+3];                   
		         }
		
			data_num_add[i].read_reference_number.x=read_len;
			data_num_add[i].read_reference_number.y=ref_len;
			data_num_add[i].address_array=data_size;

			memcpy(data_h,inputdata[i].read_base,read_len);
			data_h+=(read_len+128-1)/128*128;
			data_size+=(read_len+128-1)/128*128;

			memcpy(data_h,reference_tep,sizeof(char4)* new_len);
		        data_h+=(new_len*sizeof(char4)+127)/128*128;
		        data_size+=(new_len*sizeof(char4)+127)/128*128;
		}
		
		int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+127)/128*128;

		hipMemcpy(data_d_total,data_h_total,data_size_to_copy,hipMemcpyHostToDevice);
		NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
		char * data_d=data_d_total+(size*sizeof(NUM_ADD)+127)/128*128;
		int4 * result_d=(int4 *) (data_d_total+data_size_to_copy);
		
		char * cigar;
		hipMalloc( (char **) &cigar, size * (128* sizeof (char)+128*sizeof(int)));
		
		int * cigar_int=(int *) (cigar+size*128*sizeof(char));
		int * direction;

		hipMalloc( (int **) & direction, (size+1) * (640*1100* sizeof (int)));
	
		dim3 block(576);
		dim3 grid(size);
		clock_gettime(CLOCK_MONOTONIC_RAW,&start);
		calculate_cigar<<<grid,block>>> (size,data_d,num_add_d,result_d,direction); //result
	//	calculate_cigar_2<<<grid,block>>> (size,result_d,cigar,cigar_int,direction); //result
		hipMemcpy(result_h,result_d,size*sizeof(int)*4,hipMemcpyDeviceToHost);
		hipMemcpy(cigar_h,cigar,128*sizeof(char)*size, hipMemcpyDeviceToHost);
		hipMemcpy(cigar_int_h,cigar_int,128*sizeof(int)*size,hipMemcpyDeviceToHost);		

		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
                computation_time+=diff(start,finish);
	
	//	for(int i=0;i<size;i++)
		{
	//		printf("%d %d\n",result_h[i*4],result_h[i*4+1]);
	/*		printf("[");
			for(int j=0;j<result_h[i*4+3];j++)
			{
			if(j!=0) printf(", ");
			printf("%d%c",cigar_int_h[128*i+j],cigar_h[128*i+j]);
			}
			printf("]\n");
	*/	}

		hipFree(direction);
		free(data_h_total);
		hipFree(data_d_total);
		free(inputdata);
		hipFree(cigar);
		free(cigar_int_h);
		free(cigar_h);
 //               fscanf(file,"%d",&size);
        }

 //	printf(" computation_time= %e  total_time=%e \n",computation_time,0);
printf(" computation_time= %e  %d GCUPs=%lf\n",computation_time,total_size,( (double)total_size)/computation_time/1000000000);



        return 0;
}



