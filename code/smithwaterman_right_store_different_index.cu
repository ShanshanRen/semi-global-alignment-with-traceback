#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <unistd.h>
#include <omp.h>
#include<limits>
 double diff(timespec start, timespec end)
 {
 	double a=0;
        if((end.tv_nsec-start.tv_nsec)<0)
        {
        a=end.tv_sec-start.tv_sec-1;
        a+=(1000000000+end.tv_nsec-start.tv_nsec)/1000000000.0;
        }
        else
        {
        a=end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

        }
        return a;

}

struct NUM_ADD
{
                short2 read_reference_number;
                int address_array;
};


  __global__ void  calculate_cigar( int size, char * data, NUM_ADD *num_add,int * result, char * cigar,int * cigar_int,int * direction) //, char * result
{
	int offset=blockIdx.x;
	__shared__ short2 read_reference_number;
	__shared__ char * read_base_array;
	__shared__ char4 * reference_base_array;
	__shared__ int mismatch;
	__shared__ int  match;
	__shared__ int open;
	__shared__ int extend;
	__shared__ int * direction_index;
	__shared__ char * cigar_store;
	__shared__ int *cigar_int_store;
	while(offset<size)
         {
		if( threadIdx.x==0)
                {
                read_reference_number=num_add[offset].read_reference_number;
                read_base_array=(char *) (data+num_add[offset].address_array);
	        reference_base_array=(char4 *) (read_base_array+(read_reference_number.x+4-1)/4*4);
		direction_index=(int *) (direction+offset*800*1100);  
		cigar_store=(char *) (cigar+offset*sizeof(char)*128);
		cigar_int_store=(int *) (cigar_int+offset*128);
	     }
                __syncthreads();

 		__shared__ char reference_base_in_char[600];
                int hh=(read_reference_number.y+4-1)/4;
                int tt=(hh+blockDim.x-1)/blockDim.x;
                for(int ii=0;ii<tt;ii++)
                {
                        int aa=threadIdx.x+ii*blockDim.x;
                        if(aa< hh)
                        {
                        char4 reference_base_in_thread;
                        reference_base_in_thread=reference_base_array[aa]; //Is it right to get data from global memory
                        reference_base_in_char[aa*4]=reference_base_in_thread.x;
                        reference_base_in_char[aa*4+1]=reference_base_in_thread.y;
                        reference_base_in_char[aa*4+2]=reference_base_in_thread.z;
                        reference_base_in_char[aa*4+3]=reference_base_in_thread.w;
			 }
                }
		
		__shared__ int MM[130]; 
                __shared__ int gap_h[130]; //insertion
                __shared__ int gap_size_h[130];  //insertion
	 	__shared__ int result_col;
		__shared__ int result_row;
		__shared__ int result_col_index;
		__shared__ int result_row_index;
		__shared__ int final_result;
		__shared__ int final_i;
		__shared__ int final_j;
		__shared__ char cigar_m[128];
		__shared__ int cigar_int_m[128];

                if(threadIdx.x==0)
                {
                        MM[0]=0;
                        gap_h[0]=-1000000000;//std::numeric_limits<int>::min()/2;
                        gap_size_h[0]=0;
			match=200;
			mismatch=-150;
			open=-260;
			extend=-11;
			result_col=-1000000000;//std::numeric_limits<int>::min()/2;
			result_row=-1000000000;//std::numeric_limits<int>::min()/2;
                //	for(int i=0;i<read_reference_number.y;i++)
		//	printf("%c",reference_base_in_char[i]);
		//	printf("\n");
		//	for(int i=0;i<read_reference_number.x;i++)
		//	printf("%c",read_base_array[i]);
		}

		 __syncthreads();


		int read_number=read_reference_number.x;
                int round_size;
		int round=(read_reference_number.x+blockDim.x-1)/blockDim.x;
		for(int i=0;i<round;i++)
                {
                        round_size=(read_number>blockDim.x)?blockDim.x: read_number;
                        read_number=(read_number>blockDim.x)?read_number-blockDim.x:0; // read_num is the remaining length at this round
			
			char read_base;
			 if(threadIdx.x<round_size ) // tid is from 0 ~ round_size-1
                        {
                                read_base=read_base_array[threadIdx.x+blockDim.x*i];
			}

			int gap_v=-1000000000;//std::numeric_limits<int>::min()/2;;
			int gap_size_v=0; //Deletion
 			int M=0; //now 
                        int step_right; //now
			int ki=0;//insertion  h  negetive
			 //deletion  v
			int MMM=0; 
			int curBtrack;
                                          
			int current_reference_id=0;
                        for(int j=0;j<round_size+read_reference_number.y-1;j++)
                        {
                                int aa=j-threadIdx.x;
                                if( aa>=0 && (current_reference_id<read_reference_number.y))
                                {
					int prev_gap=M+open; //M which is cacluated by last step in the same thread
					gap_v+=extend;
					if(prev_gap>gap_v)
					{
						gap_v=prev_gap;
						gap_size_v=1;
					}	
					else
						gap_size_v++;
					
					char reference_base_each=reference_base_in_char[current_reference_id];
				
				 	current_reference_id++;
					M=MMM+(read_base==reference_base_each? match:mismatch);
					prev_gap=MM[threadIdx.x]+open;
					step_right=gap_h[threadIdx.x]+extend;
					if(prev_gap>step_right)
					{
						step_right=prev_gap;
						ki=1;
					}	
					else
						ki=gap_size_h[threadIdx.x]+1;

					bool diag=(M>=gap_v)&&(M>=step_right);
					if(diag)
					{
					  curBtrack=0;
					}
					else
					if(step_right>=gap_v)
					{
						 M=step_right;
						 curBtrack=0-ki;
					}
					else
						{
							M=gap_v;
							curBtrack=gap_size_v;
						}
					MMM=MM[threadIdx.x];
					direction_index[800*j+threadIdx.x]=curBtrack;
				
				//if(threadIdx.x==read_reference_number.x-3)
				//printf("%p %d ", &direction_index[800*j+threadIdx.x],curBtrack);
 				
				if(current_reference_id==read_reference_number.y)
				{	
					if(M>=result_row)
					{
						result_row=M;
						result_row_index=threadIdx.x;  //
					}
					//printf("%d %d  %d  %d %d \n",read_reference_number.y,M,result_row,result_row_index,threadIdx.x);
				}
                         	if(threadIdx.x==round_size-1 && i==round-1)
                                {
						if(M>=result_col)
						{
							result_col=M;
							result_col_index=current_reference_id;	// +1					
						}						
				}



			       }
                        	
				__syncthreads(); //to make sure that the former value of MM[threadIdx.x+1] are used by other threads.
                                MM[threadIdx.x+1]=M;
                                gap_h[threadIdx.x+1]=step_right;
                                gap_size_h[threadIdx.x+1]=ki;

                                                               __syncthreads(); // there should be two synthreads(); // to make sure that all of MM[threadIdx.x+1] have get a new value before M,D and I changed.
                        }
                }
		
		__shared__ int state;//0  match;  1 mistmatch; 2 inseriton;  3  deletion
		__shared__ int cigar_index;
		__shared__ int segment_length;
		__shared__ int btr;
               	__shared__ int new_state;
		__shared__ int step_length;
		 if(threadIdx.x==round_size-1)
                {
			//printf("%d %d %d %d\n", result_row,result_col, result_row_index,result_col_index);
                        if(result_row>result_col||result_row==result_col&&(read_reference_number.x-result_row_index-1)>(read_reference_number.y-result_col_index))
			{
				final_result=result_row;			
				final_i=read_reference_number.y-1;
				final_j=result_row_index;
				segment_length=read_reference_number.x-1-result_row_index;
			}
			else
			{
				final_result=result_col;
				final_i=result_col_index-1;
				final_j=read_reference_number.x-1;
				segment_length=0;
			}
			result[offset*3]=final_result;
			//printf("%d\n",final_result);
               		cigar_index=0;	
			if(segment_length>0)
			{
			cigar_m[cigar_index]='S';
			cigar_int_m[cigar_index]=segment_length;
			segment_length=0;
			cigar_index++;
			}
			
			//printf("\n %d %d\n", final_i,final_j);
			state=0;
			do
			{
				btr=direction_index[(final_i+final_j)*800+final_j];
				if(btr>0)
				{
					new_state=3;
					step_length=btr;
				}
				else
				if(btr<0)
				{
					new_state=2;
					step_length=0-btr;		
				}
				else	
				{
					new_state=0;
					step_length=1;
				}
				
		//		printf(" %d %d %d %d\n", state,new_state,final_i,final_j);
				if(new_state==0)
				{	final_i--;
					final_j--;
				}
				else
				if(new_state==2)
					final_j-=step_length;
				else
					final_i-=step_length;
			
				if(state==new_state) segment_length+=step_length;
				else
				{
				if(state==0)  cigar_m[cigar_index]='M';
				if(state==2)  cigar_m[cigar_index]='I';
				if(state==3)  cigar_m[cigar_index]='D';
                       		 cigar_int_m[cigar_index]=segment_length;
                       		 segment_length=step_length;
                       		 cigar_index++;
			 	 state=new_state;
				}
	
			}while(final_i>=0&&final_j>=0);
			
			if(state==0)  cigar_m[cigar_index]='M';
			if(state==2)  cigar_m[cigar_index]='I';
			if(state==3)  cigar_m[cigar_index]='D';
                       	cigar_int_m[cigar_index]=segment_length;
                       	cigar_index++;
			if(final_j>=0) 
			{	
				cigar_m[cigar_index]='S';
				cigar_int_m[cigar_index]=final_j+1;
				cigar_index++;
			}	

			result[offset*3+1]=final_i+1;
			result[offset*3+2]=cigar_index;
	/*		for(int i=cigar_index-1;i>=0;i--)
			{
			printf("%d%c",cigar_int_m[i],cigar_m[i]);
			}
*/
		 }
		 __syncthreads();
		if(threadIdx.x<cigar_index && cigar_index<=blockDim.x)
		{
	//	if(threadIdx.x==0)
	//		printf("%c %d\n",cigar_m[cigar_index-1-threadIdx.x], cigar_int_m[cigar_index-1-threadIdx.x]);
		cigar_store[threadIdx.x]=cigar_m[cigar_index-1-threadIdx.x];
		cigar_int_store[threadIdx.x]=cigar_int_m[cigar_index-1-threadIdx.x];
	//	if(threadIdx.x==0)
	//		printf("%c %d\n", cigar_store[threadIdx.x],cigar_int_store[threadIdx.x]);
		
		}

		offset+=gridDim.x;
	}
}

struct InputData
{
char read_base[600];
char reference_base[600];
};


int main()
{
	FILE * file;
        file=fopen("pairHMM_input.txt","r");
        int size;
        fscanf(file,"%d",&size);
	double  computation_time=0;//total_time=0;
	timespec start,finish;	 
       //printf("size=%d\n",size);
         while(!feof(file))
        {
                InputData * inputdata=(InputData* )malloc(size*(sizeof(InputData)));
                for(int i=0;i<size;i++)
                {
                        fscanf(file,"%s ",inputdata[i].reference_base);
                        fscanf(file,"%s ",inputdata[i].read_base);
                }
                
		//data preparation.
		char * data_h_total=(char*)malloc(size * 600* sizeof (char)*2);
		NUM_ADD * data_num_add=(NUM_ADD *) (data_h_total);
		char * data_h=data_h_total+size*sizeof(NUM_ADD);  // it is 64*x .thus we donot need to worry about align
		int data_size=0;
		char * data_d_total;		
		hipMalloc( (char **) &data_d_total, size * 600* sizeof (char)*2+sizeof(int)*size*3);
		int * result_h=(int *) malloc(sizeof(int)*size*3);
	 	char * cigar_h=(char *) malloc(sizeof(char)*size*128);
		int * cigar_int_h=(int *) malloc(sizeof(int)*size*128);
		for(int i=0;i<size;i++)
		{

			char4 reference_tep[150];
			int read_len=strlen(inputdata[i].read_base);
			int ref_len=strlen(inputdata[i].reference_base);
			int new_len=(ref_len+4-1)/4;
			for(int j=0;j<new_len;j++)
		        {
		        	reference_tep[j].x=inputdata[i].reference_base[j*4];
		                if(j*4+1<ref_len)
		                reference_tep[j].y=inputdata[i].reference_base[j*4+1];
		                if(j*4+2<ref_len)
		                reference_tep[j].z=inputdata[i].reference_base[j*4+2];
		                if(j*4+3<ref_len)
		                reference_tep[j].w=inputdata[i].reference_base[j*4+3];                   
		         }
		
			data_num_add[i].read_reference_number.x=read_len;
			data_num_add[i].read_reference_number.y=ref_len;
			data_num_add[i].address_array=data_size;

			memcpy(data_h,inputdata[i].read_base,read_len);
			data_h+=(read_len+4-1)/4*4;
			data_size+=(read_len+4-1)/4*4;

			memcpy(data_h,reference_tep,sizeof(char4)* new_len);
		        data_h+=new_len*sizeof(char4);
		        data_size+=new_len*sizeof(char4);
		}
		
		int data_size_to_copy=data_size+size*sizeof(NUM_ADD);

		hipMemcpy(data_d_total,data_h_total,data_size_to_copy,hipMemcpyHostToDevice);
		NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
		char * data_d=data_d_total+sizeof(NUM_ADD)*size;
		int * result_d=(int *) (data_d_total+data_size_to_copy);
		
		char * cigar;
		hipMalloc( (char **) &cigar, size * (128* sizeof (char)+128*sizeof(int)));
		
		int * cigar_int=(int *) (cigar+size*128*sizeof(char));
		int * direction;

		hipMalloc( (int **) & direction, size * (800*1100* sizeof (int)));
	
		dim3 block(128);
		dim3 grid(size);
		clock_gettime(CLOCK_MONOTONIC_RAW,&start);
		calculate_cigar<<<grid,block>>> (size,data_d,num_add_d,result_d,cigar,cigar_int,direction); //result
		hipMemcpy(result_h,result_d,size*sizeof(int)*3,hipMemcpyDeviceToHost);
		hipMemcpy(cigar_h,cigar,128*sizeof(char)*size, hipMemcpyDeviceToHost);
		hipMemcpy(cigar_int_h,cigar_int,128*sizeof(int)*size,hipMemcpyDeviceToHost);		

		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
                computation_time+=diff(start,finish);
	
		for(int i=0;i<size;i++)
		{
			printf("%d %d\n",result_h[i*3],result_h[i*3+1]);
			printf("[");
			for(int j=0;j<result_h[i*3+2];j++)
			{
			if(j!=0) printf(", ");
			printf("%d%c",cigar_int_h[128*i+j],cigar_h[128*i+j]);
			}
			printf("]\n");
		}
		hipFree(direction);
		free(data_h_total);
		hipFree(data_d_total);
		free(inputdata);
		hipFree(cigar);
		free(cigar_int_h);
		free(cigar_h);
                fscanf(file,"%d",&size);
        }

 	printf(" computation_time= %e  total_time=%e \n",computation_time,0);



        return 0;
}



