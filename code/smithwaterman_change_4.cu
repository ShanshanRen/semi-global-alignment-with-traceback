#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <unistd.h>
#include <omp.h>
#include<limits>
 double diff(timespec start, timespec end)
 {
 	double a=0;
        if((end.tv_nsec-start.tv_nsec)<0)
        {
        a=end.tv_sec-start.tv_sec-1;
        a+=(1000000000+end.tv_nsec-start.tv_nsec)/1000000000.0;
        }
        else
        {
        a=end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

        }
        return a;

}

struct NUM_ADD
{
                short2 read_reference_number;
                int address_array;
};


  __global__ void  calculate_cigar( int size, char * data, NUM_ADD *num_add,int * result, char * cigar,int * cigar_int,int * direction) //, char * result
{
	int offset=blockIdx.x;
	__shared__ short2 read_reference_number;
	__shared__ char * read_base_array;
	__shared__ char4 * reference_base_array;
	__shared__ int mismatch;
	__shared__ int  match;
	__shared__ int open;
	__shared__ int extend;
	__shared__ short2 * direction_index;
	__shared__ char * cigar_store;
	__shared__ int *cigar_int_store;
	while(offset<size)
         {
		if( threadIdx.x==0)
                {
                read_reference_number=num_add[offset].read_reference_number;
                read_base_array=(char *) (data+num_add[offset].address_array);
	        reference_base_array=(char4 *) (read_base_array+(read_reference_number.x+127)/128*128);
		direction_index=(short2 *) (direction+offset*640*1100);  
		cigar_store=(char *) (cigar+offset*sizeof(char)*128);
		cigar_int_store=(int *) (cigar_int+offset*128);
	     }
                __syncthreads();

 		__shared__ char reference_base_in_char[600];
                int hh=(read_reference_number.y+4-1)/4;
                int tt=(hh+blockDim.x-1)/blockDim.x;
                for(int ii=0;ii<tt;ii++)
                {
                        int aa=threadIdx.x+ii*blockDim.x;
                        if(aa< hh)
                        {
                        char4 reference_base_in_thread;
                        reference_base_in_thread=reference_base_array[aa]; //Is it right to get data from global memory
                        reference_base_in_char[aa*4]=reference_base_in_thread.x;
                        reference_base_in_char[aa*4+1]=reference_base_in_thread.y;
                        reference_base_in_char[aa*4+2]=reference_base_in_thread.z;
                        reference_base_in_char[aa*4+3]=reference_base_in_thread.w;
			 }
                }
		
		__shared__ int MM[130]; 
                __shared__ int gap_h[130]; //insertion
                __shared__ short2 gap_size_h[130];  //insertion
	 	__shared__ int result_col;
		__shared__ int result_row;
		__shared__ int result_col_index;
		__shared__ int result_row_index;
			__shared__ char cigar_m[128];
		__shared__ int cigar_int_m[128];
		int final_result;
	        int final_i;
	        int final_j;


                if(threadIdx.x==0)
                {
                        MM[0]=0;
                        gap_h[0]=-1000000000;//std::numeric_limits<int>::min()/2;
                        gap_size_h[0].x=0;
			gap_size_h[0].y=0;
			match=200;
			mismatch=-150;
			open=-260;
			extend=-11;
			result_col=-1000000000;//std::numeric_limits<int>::min()/2;
			result_row=-1000000000;//std::numeric_limits<int>::min()/2;
                //	for(int i=0;i<read_reference_number.y;i++)
		//	printf("%c",reference_base_in_char[i]);
		//	printf("\n");
		//	for(int i=0;i<read_reference_number.x;i++)
		//	printf("%c",read_base_array[i]);
		}

		 __syncthreads();


		int read_number=read_reference_number.x;
                {
			
			char read_base;
                        read_base=read_base_array[threadIdx.x];

			int gap_v=-1000000000;//std::numeric_limits<int>::min()/2;;
			int gap_size_v=0; //Deletion
 			int M=0; //now 
                        int step_right; //now
			int ki=0;//insertion  h  negetive
			 //deletion  v
			int MMM=0; 
                        short mt=0; 
			short2 curmt;
			curmt.x=0;
			curmt.y=0;               
			int current_reference_id=0;
                        for(int j=0;j<read_reference_number.x+read_reference_number.y-1;j++)
                        {
                                int aa=j-threadIdx.x;
                                if( aa>=0 && (current_reference_id<read_reference_number.y))
                                {
					int prev_gap=M+open; //M which is cacluated by last step in the same thread
					gap_v+=extend;
					if(prev_gap>gap_v)
					{
						gap_v=prev_gap;
						gap_size_v=1;
					}	
					else
						gap_size_v++;
					
					char reference_base_each=reference_base_in_char[current_reference_id];
				
					M=MMM+(read_base==reference_base_each? match:mismatch);
					prev_gap=MM[threadIdx.x]+open;
					step_right=gap_h[threadIdx.x]+extend;
					if(prev_gap>step_right)
					{
						step_right=prev_gap;
						ki=1;
					}	
					else
						ki=gap_size_h[threadIdx.x].x+1;

					bool diag=(M>=gap_v)&&(M>=step_right);
					curmt.y=0;
				
					if(diag)
					{
					curmt.x=0;
					//if(threadIdx.x==0||current_reference_id==0)
					//	curmt.y=0;
				//	else
						curmt.y=mt+1;
					//  curBtrack=0;
					}
					else
					if(step_right>=gap_v)
					{
						 M=step_right;
						curmt.x=0-ki;
						// curBtrack=0-ki;
					}
					else
						{
							M=gap_v;
							curmt.x=gap_size_v;
							//curBtrack=gap_size_v;
						}
					MMM=MM[threadIdx.x];
					mt=gap_size_h[threadIdx.x].y;
					direction_index[640*j+threadIdx.x]=curmt;
			//if(threadIdx.x==read_reference_number.x-3)
				//printf("%p %d ", &direction_index[800*j+threadIdx.x],curBtrack);
 				
				if(current_reference_id==read_reference_number.y-1)
				{	
					if(M>=result_row)
					{
						result_row=M;
						result_row_index=threadIdx.x;  //
					}
					//printf("%d %d  %d  %d %d \n",read_reference_number.y,M,result_row,result_row_index,threadIdx.x);
				}
                         	if(threadIdx.x==read_reference_number.x-1)
                                {
						if(M>=result_col)
						{
							result_col=M;
							result_col_index=current_reference_id;	// +1					
						}						
				}

				current_reference_id++;
		
			//	if(threadIdx.x==5)
			//		printf("%d  ", curmt.y);

			       }
                        	
				__syncthreads(); //to make sure that the former value of MM[threadIdx.x+1] are used by other threads.
                                MM[threadIdx.x+1]=M;
                                gap_h[threadIdx.x+1]=step_right;
                                gap_size_h[threadIdx.x+1].x=ki;
				gap_size_h[threadIdx.x+1].y=curmt.y;
                                __syncthreads(); // there should be two synthreads(); // to make sure that all of MM[threadIdx.x+1] have get a new value before M,D and I changed.
                        }
                }
			char state;//0  match;  1 mistmatch; 2 inseriton;  3  deletion
		__shared__ int cigar_index;
		int segment_length;
		short2 btr;
               	char  new_state;
		int step_length;
		 if(threadIdx.x==read_reference_number.x-1)
                {
			//printf("%d %d %d %d\n", result_row,result_col, result_row_index,result_col_index);
                        if(result_row>result_col||result_row==result_col&&(read_reference_number.x-result_row_index-1)>(read_reference_number.y-result_col_index-1))
			{
				final_result=result_row;			
				final_i=read_reference_number.y-1;
				final_j=result_row_index;
				segment_length=read_reference_number.x-1-result_row_index;
			}
			else
			{
				final_result=result_col;
				final_i=result_col_index;
				final_j=read_reference_number.x-1;
				segment_length=0;
			}
			result[offset*3]=final_result;
			//printf("%d\n",final_result);
               		cigar_index=0;	
			if(segment_length>0)
			{
			cigar_m[cigar_index]='S';
			cigar_int_m[cigar_index]=segment_length;
			segment_length=0;
			cigar_index++;
			}
			
			//printf("\n %d %d\n", final_i,final_j);
			//state=4;
			state='N';
			do
			{
				btr=direction_index[(final_i+final_j)*640+final_j];
				if(btr.x>0)
				{
					new_state='D';
				//	new_state=3;
					step_length=btr.x;
					final_i-=step_length;
				}
				else
				if(btr.x<0)
				{
					new_state='I';
				//	new_state=2;
					step_length=0-btr.x;		
					final_j-=step_length;
				}
				else	
				{
					new_state='M';
				//	new_state=0;
					//if(btr.y==0)
					//step_length=1;
					//else
					step_length=btr.y;
					final_i-=step_length;
					final_j-=step_length;
			
				}
				
		//	printf(" %d %d %d %d\n", state,new_state,final_i,final_j);
		/*		if(new_state==0)
				{	final_i-=step_length;
					final_j-=step_length;
				}
				else
				if(new_state==2)
					final_j-=step_length;
				else
					final_i-=step_length;
		*/	
				//if(state==4) state=new_state;
				if(state=='N') state=new_state;
				if(state==new_state) 
				{
					segment_length+=step_length;
				}
				else
				{
			//	if(state==0) cigar_m[cigar_index]='M';
			//	if(state==2) cigar_m[cigar_index]='I';
			//	if(state==3) cigar_m[cigar_index]='D';
				 cigar_m[cigar_index]=state;
                       		 cigar_int_m[cigar_index]=segment_length;
                       		 segment_length=step_length;
                       		 cigar_index++;
			 	 state=new_state;
				}
	
			}while(final_i>=0&&final_j>=0);
				//if(state==0) cigar_m[cigar_index]='M';
				//if(state==2) cigar_m[cigar_index]='I';
				//if(state==3) cigar_m[cigar_index]='D';
                       	
			cigar_m[cigar_index]=state;
                       	cigar_int_m[cigar_index]=segment_length;
                       	cigar_index++;
			if(final_j>=0) 
			{	
				cigar_m[cigar_index]='S';
				cigar_int_m[cigar_index]=final_j+1;
				cigar_index++;
			}	

			result[offset*3+1]=final_i+1;
			result[offset*3+2]=cigar_index;
	/*		for(int i=cigar_index-1;i>=0;i--)
			{
			printf("%d%c",cigar_int_m[i],cigar_m[i]);
			}
*/
		 }
		 __syncthreads();
		if(threadIdx.x<cigar_index && cigar_index<=blockDim.x)
		{
	//	if(threadIdx.x==0)
	//		printf("%c %d\n",cigar_m[cigar_index-1-threadIdx.x], cigar_int_m[cigar_index-1-threadIdx.x]);
		cigar_store[threadIdx.x]=cigar_m[cigar_index-1-threadIdx.x];
		cigar_int_store[threadIdx.x]=cigar_int_m[cigar_index-1-threadIdx.x];
	//	if(threadIdx.x==0)
	//		printf("%c %d\n", cigar_store[threadIdx.x],cigar_int_store[threadIdx.x]);
		
		}

		offset+=gridDim.x;
	}
}

struct InputData
{
char read_base[600];
char reference_base[600];
};


int main(int artc, char* args[])
{
	FILE * file;
        file=fopen(args[1],"r");
        int size;
     //   fscanf(file,"%d",&size);
	double  computation_time=0;//total_time=0;
	timespec start,finish;	 
     
	/*  char data[200][1000];
                for(int i=0;i<101;i++)
                {
                        fscanf(file,"%s ", data[i]);
                }
                int row=atoi(args[2]);
                int col=atoi(args[3]);
                size=row*col;
        for(int ww=0;ww<1;ww++)
        {       int index=0;
                InputData * inputdata=(InputData* )malloc(size*(sizeof(InputData)));
                for(int i=0;i<row;i++)
                for(int j=0;j<col;j++)
                {
                        strcpy(inputdata[index].reference_base,data[1]);
                        strcpy(inputdata[index].read_base,data[1]);
                        index++;
                }       
	*/


               
		//data preparation.
		char * data_h_total=(char*)malloc(size * 640* sizeof (char)*2);
		NUM_ADD * data_num_add=(NUM_ADD *) (data_h_total);
		char * data_h=data_h_total+(size*sizeof(NUM_ADD)+127)/128*128;  // it is 64*x .thus we donot need to worry about align
		int data_size=0;
		char * data_d_total;		
		hipMalloc( (char **) &data_d_total, (size*sizeof(NUM_ADD)+127)/128*128+size *( 640 )* sizeof (char)*2+sizeof(int)*size*3);
		int * result_h=(int *) malloc(sizeof(int)*size*3);
	 	char * cigar_h=(char *) malloc(sizeof(char)*size*128);
		int * cigar_int_h=(int *) malloc(sizeof(int)*size*128);
		for(int i=0;i<size;i++)
		{

			char4 reference_tep[150];
			int read_len=strlen(inputdata[i].read_base);
			int ref_len=strlen(inputdata[i].reference_base);
			int new_len=(ref_len+4-1)/4;
			for(int j=0;j<new_len;j++)
		        {
		        	reference_tep[j].x=inputdata[i].reference_base[j*4];
		                if(j*4+1<ref_len)
		                reference_tep[j].y=inputdata[i].reference_base[j*4+1];
		                if(j*4+2<ref_len)
		                reference_tep[j].z=inputdata[i].reference_base[j*4+2];
		                if(j*4+3<ref_len)
		                reference_tep[j].w=inputdata[i].reference_base[j*4+3];                   
		         }
		
			data_num_add[i].read_reference_number.x=read_len;
			data_num_add[i].read_reference_number.y=ref_len;
			data_num_add[i].address_array=data_size;

			memcpy(data_h,inputdata[i].read_base,read_len);
			data_h+=(read_len+128-1)/128*128;
			data_size+=(read_len+128-1)/128*128;

			memcpy(data_h,reference_tep,sizeof(char4)* new_len);
		        data_h+=(new_len*sizeof(char4)+127)/128*128;
		        data_size+=(new_len*sizeof(char4)+127)/128*128;
		}
		
		int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+127)/128*128;

		hipMemcpy(data_d_total,data_h_total,data_size_to_copy,hipMemcpyHostToDevice);
		NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
		char * data_d=data_d_total+(size*sizeof(NUM_ADD)+127)/128*128;
		int * result_d=(int *) (data_d_total+data_size_to_copy);
		
		char * cigar;
		hipMalloc( (char **) &cigar, size * (128* sizeof (char)+128*sizeof(int)));
		
		int * cigar_int=(int *) (cigar+size*128*sizeof(char));
		int * direction;

		hipMalloc( (int **) & direction, size * (640*1100* sizeof (int)));
	
		dim3 block(128);
		dim3 grid(size);
		clock_gettime(CLOCK_MONOTONIC_RAW,&start);
		calculate_cigar<<<grid,block>>> (size,data_d,num_add_d,result_d,cigar,cigar_int,direction); //result
		hipMemcpy(result_h,result_d,size*sizeof(int)*3,hipMemcpyDeviceToHost);
		hipMemcpy(cigar_h,cigar,128*sizeof(char)*size, hipMemcpyDeviceToHost);
		hipMemcpy(cigar_int_h,cigar_int,128*sizeof(int)*size,hipMemcpyDeviceToHost);		

		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
                computation_time+=diff(start,finish);
	
		for(int i=0;i<size;i++)
		{
			printf("%d %d\n",result_h[i*3],result_h[i*3+1]);
			printf("[");
			for(int j=0;j<result_h[i*3+2];j++)
			{
			if(j!=0) printf(", ");
			printf("%d%c",cigar_int_h[128*i+j],cigar_h[128*i+j]);
			}
			printf("]\n");
		}

		hipFree(direction);
		free(data_h_total);
		hipFree(data_d_total);
		free(inputdata);
		hipFree(cigar);
		free(cigar_int_h);
		free(cigar_h);
 //               fscanf(file,"%d",&size);
        }

 	printf(" computation_time= %e  total_time=%e \n",computation_time,0);



        return 0;
}



