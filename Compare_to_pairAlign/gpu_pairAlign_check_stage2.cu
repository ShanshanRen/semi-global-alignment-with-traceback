#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <unistd.h>
#include <omp.h>
#include<limits>

#define MEM_OFFSET gridDim.x*blockDim.x  //the number of threads in the grid
#define BACK(x,y)   back[startPosA[blockThread] + ( ( ((y) + 8) / 8) * rowWidth + (x) + 1 ) * MEM_OFFSET]
#define Y_STEPS 8
#define BLOCK_SIZE   128   //128   //blockDim.x
#define INT_INT -2147483647

 double diff(timespec start, timespec end)
 {
 	double a=0;
        if((end.tv_nsec-start.tv_nsec)<0)
        {
        a=end.tv_sec-start.tv_sec-1;
        a+=(1000000000+end.tv_nsec-start.tv_nsec)/1000000000.0;
        }
        else
        {
        a=end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

        }
        return a;

}

struct NUM_ADD
{
                short2 read_reference_number;
                int address_array;
};


//calculate_cigar( int size, char * data, NUM_ADD *num_add,int4 * result, int * direction) 
//rowWidth=640

__global__ void Calculate_ben(int size, char *data, NUM_ADD * num_add, short4 *result, int2* AF_maxXY, unsigned int* back,  short rowWidth)
{
	int offset=blockIdx.x*blockDim.x+threadIdx.x;
	short2 lengthXY;
	char * read_base_array;
	char4 * reference_base_array;
	int  mismatch;
	int  match;
	int gapOp;
	int  gapEx;
	__shared__  int startPosA[128];  //blockDim.x
	while(offset<size)
	{
		int blockThread=threadIdx.x;
		//if(offset>=2943) printf("id=%d\n",offset);
		match=200;
		mismatch=-150;
		gapOp=-260;
		gapEx=-11;
		lengthXY=num_add[offset].read_reference_number;
		read_base_array=(char *) (data+num_add[offset].address_array);
		reference_base_array=(char4 *) (read_base_array+(lengthXY.x+127)/128*128);
		// direction_index=(short2 *)(direction+offset*640*1100);  
		startPosA[threadIdx.x] = offset;  // startPosA 是放的是什么？
		//startPosA 在原程序中是代表 thread在grid中的编号
		//	printf("%d %d\n", lengthXY.x, lengthXY.y);

		//下面是初始化

		//initialization of the -1 row in A matrix
		// - 2 bytes for element of A matrix
		// - 2 bytes for element of F matrix
		//还是不知道AF_maxXY 是放什么的？
		for(short x = 0; x < lengthXY.x; x++)
		{
			int2 tmp;
			//(x + 1) because the first element should be -gapEx
			tmp.x = 0;
			tmp.y = INT_INT - gapEx;
			AF_maxXY[startPosA[threadIdx.x] + x * MEM_OFFSET ] = tmp;  //because of this operation, the total number of threads in the grid should be greater than size. Otherwise, AF would re-written.
			//MEM_OFFSET在原程序中是 grid的总的thread的个数。
			//AF_maxXY[startPosA[blockThread] + x * MEM_OFFSET] = tmp;

			//fill the -1 row of "back" array
			BACK(x,-1) = 9; //0000 0000 0000 0000 0000 0000 0000 1001 == 9
		}

	  //	if(offset>=2943) printf("id=%d\n",offset);
		//	printf("%d %d\n", lengthXY.x, lengthXY.y);
		//fill the -1 column of "back" array
		for(short y = 0; y < lengthXY.y; y+=Y_STEPS)
		{
			
	//	if(offset>=2943) printf("id=%d %d  %d\n",offset,y,startPosA[threadIdx.x] + ( ( ((y) + 8) / 8) * rowWidth + (-1) + 1 ) * MEM_OFFSET);
			BACK(-1,y) = 1717986918; //0110 0110 0110 0110 0110 0110 0110 0110 = 1717986918
		}
		BACK(-1,-1) = 0; //stop element

		
		//one element of AE_shared consist of:
		// - one A element
		// - one E element
		__shared__ int2 AE_shared[Y_STEPS][BLOCK_SIZE];
		//elements of Y sequence go to sharedYSeq
		__shared__ char4 sharedYSeq[Y_STEPS/4][BLOCK_SIZE];


		int2 AF_current;
		AF_current.x = 0;

		__shared__ int2 ymin_score[BLOCK_SIZE]; //stores ymin and score
		ymin_score[threadIdx.x].y = 0;

		__shared__ short4 maxXY[BLOCK_SIZE];
		maxXY[threadIdx.x].x = lengthXY.x - 1;
		maxXY[threadIdx.x].y = 0;
		maxXY[threadIdx.x].z = 0;

		// |
		// |
		// |
		// V
		for (short y = 0; y < lengthXY.y; y += Y_STEPS)
		{
			//printf("%d\n",y);
			int2 A_init_upleft;
			A_init_upleft.x = 0;

			//initialialization of the -1 column in A matrix
			// - one element of A matrix
			// - one element of E matrix
			for (short i = 0; i < Y_STEPS; i++)
			{
				int2 tmp;
				tmp.x = 0;
				tmp.y = INT_INT - gapEx;
				AE_shared[i][threadIdx.x] = tmp;
			}


			//we read elements of the Y sequence
			for (short i = 0; i < Y_STEPS/4; i++)
			{
				sharedYSeq[i][threadIdx.x] = reference_base_array[y/4+i];
				//PACK_BYTES(tex1Dfetch(texSeqsY, startY + y + i*4 + 0),
				//                                       tex1Dfetch(texSeqsY, startY + y + i*4 + 1),
				//                                      tex1Dfetch(texSeqsY, startY + y + i*4 + 2),
				//                                     tex1Dfetch(texSeqsY, startY + y + i*4 + 3));
				//printf("%c %c %c %c\n", sharedYSeq[i][threadIdx.x].x,sharedYSeq[i][threadIdx.x].y,sharedYSeq[i][threadIdx.x].z,sharedYSeq[i][threadIdx.x].w);
			}

			ymin_score[threadIdx.x].x = min(Y_STEPS, lengthXY.y - y); //(i < Y_STEPS) && (i + y < lengthY)

			//------>
			for (short x = 0; x < lengthXY.x; x++)
			{
				//actual up_left gets a value of recent read value from the global memory
				//and actual read value is stored in first two bites of A_upleft
				A_init_upleft.y = A_init_upleft.x;

				char2 XYSeq;
				XYSeq.x = read_base_array[x];
				//	if(y==0) printf("%c\n",XYSeq.x);
				//read from global memory
				int2 AF_up = AF_maxXY[startPosA[threadIdx.x] + x * MEM_OFFSET];

				//A_init -> up element read in previous iteration from global memory (up-left)
				A_init_upleft.x = AF_up.x;

				int2 AE_left;
				int E_current;
				int similarity;
				unsigned int back8 = 0;
				//  |  /|  /|
				//  | / | / |
				//  |/  |/  V
				//  |  /|  /|
				//  | / | / |
				//  |/  |/  V
				for(short i = 0; i < ymin_score[threadIdx.x].x; i++)
				{
					AE_left = AE_shared[i][threadIdx.x];


					// XYSeq.y = sharedYSeq[i/4][threadIdx.x].x,y,z,w;
					if(i%4==0)
						XYSeq.y = sharedYSeq[i/4][threadIdx.x].x;
					if(i%4==1)
						XYSeq.y = sharedYSeq[i/4][threadIdx.x].y;
					if(i%4==2)
						XYSeq.y = sharedYSeq[i/4][threadIdx.x].z;
					if(i%4==3)
						XYSeq.y = sharedYSeq[i/4][threadIdx.x].w;
					//(sharedYSeq[i/4][threadIdx.x] >> (((15-i)%4) * 8)) & 0xFF;


					//similarity = substitutionMatrix[XYSeq.y*lettersCount + XYSeq.x];
					similarity =   (XYSeq.x==XYSeq.y? match:mismatch);
					similarity += A_init_upleft.y;

					E_current = max(AE_left.y + gapEx, AE_left.x + gapOp);
					AF_current.y = max(AF_up.y + gapEx, AF_up.x + gapOp);

					AF_current.x = max(E_current, AF_current.y);
					AF_current.x = max(AF_current.x, similarity);

					//"back" array
					back8 <<= 1;
					//back8 |= ((AF_current.x==E_current) && (AF_current.x!=AF_current.y)) || (AF_current.x==similarity); //if go left
					back8 |= (AF_current.x==E_current)  || (AF_current.x==similarity); //if go left
					back8 <<= 1;
					//back8 |= (AF_current.x==AF_current.y) || (AF_current.x==similarity); //if go up
					back8 |=( (AF_current.x==AF_current.y)&& (AF_current.x!=E_current)) || (AF_current.x==similarity); //if go up
					back8 <<= 1;
					back8 |= (AF_current.y == (AF_up.y + gapEx)); //if continue up
					back8 <<= 1;
					back8 |= (E_current == (AE_left.y + gapEx)); //if continue left

					//initialize variables for next iterations
					int2 AE_tmp;
					AE_tmp.x = AF_current.x;
					AE_tmp.y = E_current;
					AE_shared[i][threadIdx.x] = AE_tmp;
					A_init_upleft.y = AE_left.x;
					AF_up = AF_current;
					//	printf("%d ",AF_current.x);
				}  //end of i
				//printf("\n");
				//we want the last row of back8 to be completed
				back8 <<= 4 * (Y_STEPS - ymin_score[threadIdx.x].x);

				//write variables to global memory for next loop
				AF_maxXY[startPosA[threadIdx.x] + x * MEM_OFFSET] = AF_current;
				BACK(x,y) = back8;

				//looking for max element in the last row
				if( (y + ymin_score[threadIdx.x].x) == lengthXY.y )
				{
					if (AF_current.x > ymin_score[threadIdx.x].y)
					{
						maxXY[threadIdx.x].x = x;
						maxXY[threadIdx.x].y = y + ymin_score[threadIdx.x].x - 1; //why minus 1???? Because 0+8=8,it should be 7.
						maxXY[threadIdx.x].z=lengthXY.x-1-x;
					}
					//if result== last row
					//result4.x=read_reference_number.y-1;
					//result4.y=result_row_index;///result_row_index is the threadIdx.x,which is x.
					//result4.z=read_reference_number.x-1-result_row_index;
					ymin_score[threadIdx.x].y = max(ymin_score[threadIdx.x].y, AF_current.x);
				}

			} //end of x

			//looking for max element in the last column
			for(short i = 0; i < ymin_score[threadIdx.x].x; i++)
			{
				if (AE_shared[i][threadIdx.x].x > ymin_score[threadIdx.x].y||AE_shared[i][threadIdx.x].x==ymin_score[threadIdx.x].y&& maxXY[threadIdx.x].z>(lengthXY.y-(y+i)-1))
				{
					maxXY[threadIdx.x].x = lengthXY.x - 1; //
					maxXY[threadIdx.x].y = y + i;
					maxXY[threadIdx.x].z=0;
				}
				//result4.x=result_col_index;   //result_col_index is the y.
				//result4.y=read_reference_number.x-1;
				//result4.z=0;
				ymin_score[threadIdx.x].y = max(ymin_score[threadIdx.x].y, AE_shared[i][threadIdx.x].x);
			}
		}//end of y
		//        maxXY[threadIdx.x].w=ymin_score[threadIdx.x].y;
		//here write result (AF_current) to global memory
		//   scores[startPosA[blockThread]] = ymin_score[blockThread].y;
		// AF_maxXY[startPosA[threadIdx.x]] = maxXY[threadIdx.x];
		result[offset]=maxXY[threadIdx.x];
		//	printf("%d %d %d %d %d\n",offset,result[offset].x,result[offset].y,result[offset].z,result[offset].w);

		offset+=gridDim.x*blockDim.x;
	}


}

#undef BACK
#define BACK(x,y)   back[startPosA + ( ( ((y) + 8) / 8) * rowWidth + (x) + 1 ) * MEM_OFFSET]

#define STOP         0
#define UP           4
#define LEFT         8
#define CROSSWISE   12
#define DIRECTION   12
#define CONTIN_UP    2
#define CONTIN_LEFT  1
#define ELEMENT     15
#define ININTI      3

//calculate_cigar_2( int size, int4 * result, char * cigar,int * cigar_int,int * direction) //,
__global__ void Calculate_ben_back(int size, short4 * result, char * cigar, int *cigar_int,  unsigned int* back, short rowWidth)
{

	int offset=blockIdx.x*blockDim.x+threadIdx.x;
	while(offset<size)
	{

		char * cigar_store;
		int * cigar_int_store;
		cigar_store=(char *) (cigar+offset*sizeof(char)*128);
		cigar_int_store=(int *) (cigar_int+offset*128);
		int segment_length;

		//startPosA == thread number within whole grid   
		int startPosA = offset;

		short4 myMaxXY = result[startPosA];
		short2 indexXY;
		indexXY.x=myMaxXY.x;
		indexXY.y=myMaxXY.y;

		int cigar_index=0;	
		if(myMaxXY.z>0)
		{
			cigar_store[cigar_index]='S';
			cigar_int_store[cigar_index]=myMaxXY.z;
			cigar_index++;
		}
		segment_length=0;


		unsigned int back8 = BACK(indexXY.x, indexXY.y);
		back8 >>= ((8 - ((indexXY.y + 1) % 8)) % 8) * 4;

		unsigned char back1 = back8 & ELEMENT; //current element of back array
		back8 >>= 4;

		unsigned char prevDirection = ININTI;// 1100 == 12 =>crosswise
		unsigned todo;
		//back 1 is the current element of back array
		while(indexXY.x>=0 && indexXY.y>=0)//(back1 & DIRECTION) //while(direction != STOP)
		{

			if( ((prevDirection & DIRECTION) == UP) && (prevDirection & CONTIN_UP) )
			{
				todo = UP;
			}
			else if( ((prevDirection & DIRECTION) == LEFT) && (prevDirection & CONTIN_LEFT) )
			{
				todo = LEFT;
			}
			else if ((back1 & DIRECTION) == UP)
			{
				todo = UP;
			}
			else if ((back1 & DIRECTION) == LEFT)
			{
				todo = LEFT;
			}
			else //if (back1 & DIRECTION == CROSSWISE)
			{
				todo = CROSSWISE;
			}

			if(prevDirection==ININTI) prevDirection=todo;
			if((prevDirection & DIRECTION)==todo) 
			{
				segment_length++;
			}
			else
			{
				//printf("             prevDirectio= %d  todo=%d\n",prevDirection,todo);
				//if(prevDirection==LEFT);
				cigar_store[cigar_index]=(prevDirection & DIRECTION);//'D';  //I D M????????
				//if(prevDirection==UP)
			//	cigar_store[cigar_index]=UP;//'I';  //I D M????????
				//if(prevDirection==CROSSWISE)
			//	cigar_store[cigar_index]=CROSSWISE;//'M';  //I D M????????

				cigar_int_store[cigar_index]=segment_length;
				cigar_index++;
				segment_length=1;
				prevDirection=todo;
			}

			if (todo == LEFT)
			{
				indexXY.x--;
				back8 = BACK(indexXY.x, indexXY.y);
				back8 >>= ((8 - ((indexXY.y + 1) % 8)) % 8) * 4; //because of the last row of back array
			}
			else if (todo == UP)
			{
				indexXY.y--;
				if((indexXY.y % 8) == 7)
					back8 = BACK(indexXY.x, indexXY.y);  //since up direction, 8 elements stored in the same int.
			}
			else //if (todo == CROSSWISE)
			{
				indexXY.x--;
				indexXY.y--;

				back8 = BACK(indexXY.x, indexXY.y);
				back8 >>= ((8 - ((indexXY.y + 1) % 8)) % 8) * 4; //because of the last row of back array
			}

			prevDirection = todo | back1&3;   //Here, back1 is used to calculate preDirection.
			//printf("prevDirection=%d  %d %d \n",prevDirection,indexXY.x,indexXY.y);
			back1 = back8 & ELEMENT;
			back8 >>= 4;
		}

		//maybe S
		//**********

		cigar_store[cigar_index]=todo;
		cigar_int_store[cigar_index]=segment_length;
		cigar_index++;
	//	printf("%d\n",cigar_index);
		if(indexXY.x>=0) 
		{	
			cigar_store[cigar_index]='S';
			cigar_int_store[cigar_index]=indexXY.x+1;
			cigar_index++;
		}	

		myMaxXY.z=indexXY.x+1;
		myMaxXY.w=cigar_index;
		result[offset]=myMaxXY;
		offset+=blockDim.x*gridDim.x;
	}

}


struct InputData
{
	char read_base[600];
	char reference_base[600];
};


int main(int artc, char* args[])
{
	uint64_t total_size=0;
	FILE * file;
        file=fopen(args[1],"r");
        int size;
	double  computation_time=0;//total_time=0;
	timespec start,finish;	 

	 char data[200][1000];   //* Here, we read in 100 sequences.
     for(int i=0;i<1;i++)
     {
        fscanf(file,"%s ", data[i]);
 }
      int row=atoi(args[2]);
      int col=atoi(args[3]);
      size=row*col;
      for(int ww=0;ww<1;ww++)
      {       
      	//Here is the sequences pairs.
      	int index=0;
        InputData * inputdata=(InputData* )malloc(100*(sizeof(InputData)));
        for(int i=0;i<1;i++)
        for(int j=0;j<1;j++)
        {
        strcpy(inputdata[index].reference_base,data[i]);
        strcpy(inputdata[index].read_base,data[j]);
  
//	printf("%s\n",inputdata[index].reference_base);
//	printf("%s\n",inputdata[index].read_base);
      index++;
        }

	 for(int j=1;j<99;j++)
        {
        strcpy(inputdata[j].reference_base,inputdata[0].reference_base);
        strcpy(inputdata[j].read_base,inputdata[0].read_base);
	}
       size=100;
		//data preparation.
		//we put all the sequence pairs into a char* array
		char * data_h_total=(char*)malloc(size * 640* sizeof (char)*2+(size*sizeof(NUM_ADD)+127)/128*128);
		NUM_ADD * data_num_add=(NUM_ADD *) (data_h_total);
		char * data_h=data_h_total+(size*sizeof(NUM_ADD)+127)/128*128;  //.thus we donot need to worry about align
		int data_size=0;
		char * data_d_total;		
		hipMalloc( (char **) &data_d_total, (size*sizeof(NUM_ADD)+127)/128*128+size *( 640 )* sizeof (char)*2+sizeof(int)*size*4);
	//	printf("total size=%d\n",(size*sizeof(NUM_ADD)+127)/128*128+size *( 640 )* sizeof (char)*2+sizeof(int)*size*4);

		short * result_h=(short*) malloc(sizeof(short)*size*4);
	 //	printf("%d\n",sizeof(short)*size*4);
		char * cigar_h=(char *) malloc(sizeof(char)*size*128);   //Here the length of alignment is 128
		int * cigar_int_h=(int *) malloc(sizeof(int)*size*128);  //Here the length of alignment is 128

		for(int i=0;i<size;i++)
		{
			char4 reference_tep[150];
			int read_len=strlen(inputdata[i].read_base);
			int ref_len=strlen(inputdata[i].reference_base);
			int new_len=(ref_len+4-1)/4;
			total_size+=ref_len*read_len; 
			//printf("i=%d total_size=%d",i,total_size);
			for(int j=0;j<new_len;j++)
		        {
		        	reference_tep[j].x=inputdata[i].reference_base[j*4];
		                if(j*4+1<ref_len)
		                reference_tep[j].y=inputdata[i].reference_base[j*4+1];
		                if(j*4+2<ref_len)
		                reference_tep[j].z=inputdata[i].reference_base[j*4+2];
		                if(j*4+3<ref_len)
		                reference_tep[j].w=inputdata[i].reference_base[j*4+3];                   
		         }
		
			data_num_add[i].read_reference_number.x=read_len;
			data_num_add[i].read_reference_number.y=ref_len;
			data_num_add[i].address_array=data_size;

			memcpy(data_h,inputdata[i].read_base,read_len);
			data_h+=(read_len+128-1)/128*128;
			data_size+=(read_len+128-1)/128*128;

			memcpy(data_h,reference_tep,sizeof(char4)* new_len);
		        data_h+=(new_len*sizeof(char4)+127)/128*128;
		        data_size+=(new_len*sizeof(char4)+127)/128*128;
		}
		
		hipError_t err;		
		int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+127)/128*128;

		hipMemcpy(data_d_total,data_h_total,data_size_to_copy,hipMemcpyHostToDevice);
		NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
		char * data_d=data_d_total+(size*sizeof(NUM_ADD)+127)/128*128;
		short4 * result_d=(short4 *) (data_d_total+data_size_to_copy);
		 //printf("data size to copy=%d\n",data_size_to_copy);
		int blocksize=128;
		dim3 block(blocksize);
		dim3 grid((size+blocksize-1)/blocksize); //size/blocksize


		char * cigar;
		err=hipMalloc( (char **) &cigar, size * (128* sizeof (char)+128*sizeof(int)));
	       if (err != hipSuccess)
        {
printf("1   1 1 1 %s", hipGetErrorString(err));
        }			
		int * cigar_int=(int *) (cigar+size*128*sizeof(char));
		unsigned int * direction;
		int2 * AF_maxXY;
		err=hipMalloc((int2 **)& AF_maxXY, 640*sizeof(int2)*(size+blocksize-1)/blocksize*blocksize);// vector
		       if (err != hipSuccess)
        {
	printf("2    23     %s", hipGetErrorString(err));
        }


	//	cudaMalloc( (unsigned int **) & direction, size * (640*640* sizeof (unsigned int)));
	//	cudaMalloc( (unsigned int **) & direction, 640*640* sizeof (unsigned int)*(size+blocksize-1)/blocksize*blocksize);
		err=hipMalloc( (unsigned int **) & direction, 640*(640/8)* sizeof (unsigned int)*(size+blocksize-1)/blocksize*blocksize);
		if (err != hipSuccess)
	{
	
	printf("3      %s", hipGetErrorString(err));
	}
		
		Calculate_ben<<<grid,block>>> (size,data_d,num_add_d,result_d,AF_maxXY, direction, 640); //result
       //Calculate_ben(int size, char *data, NUM_ADD * num_add, short4 *result, int2* AF_maxXY, unsigned int* back,  short rowWidth)
		hipDeviceSynchronize();
       // Calculate_ben_back(int size, short4 * result, char * cigar, int *cigar_int,  unsigned int* back, short rowWidth)
		clock_gettime(CLOCK_MONOTONIC_RAW,&start);
	  Calculate_ben_back<<<grid,block>>> (size,result_d,cigar,cigar_int,direction,640); //result

	//	printf("%d\n", size*sizeof(short4));
		hipDeviceSynchronize();
	//	cudaMemcpy(result_h,result_d,size*sizeof(short4),cudaMemcpyDeviceToHost);
	//	cudaMemcpy(cigar_h,cigar,128*sizeof(char)*size, cudaMemcpyDeviceToHost);
	//	cudaMemcpy(cigar_int_h,cigar_int,128*sizeof(int)*size,cudaMemcpyDeviceToHost);		
		
	clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
        computation_time+=diff(start,finish);
	
/*		for(int i=0;i<size;i++)
		{
	//		printf("%d\n",result_h[i*4]);
	//		printf("%d\n",result_h[i*4+1]);
	//		printf("%d\n",result_h[i*4+2]);
	//		printf("%d\n",result_h[i*4+3]);
			printf("[");
			for(int j=result_h[i*4+3]-1;j>=0;j--)
			{
			printf("%d",cigar_int_h[128*i+j]);
			if(cigar_h[128*i+j]==UP)
			printf("%c",'D');
			if(cigar_h[128*i+j]==LEFT)
			printf("%c",'I');
			if(cigar_h[128*i+j]==CROSSWISE)
			printf("%c",'M');			
			if(cigar_h[128*i+j]=='S')
			printf("%c",'S');			
			if(j!=0) printf(", ");
			}
			printf("]\n");
		}
*/
        hipFree(AF_maxXY);
		hipFree(direction);
		free(data_h_total);
		hipFree(data_d_total);
		free(inputdata);
		hipFree(cigar);
		free(cigar_int_h);
		free(cigar_h);
 //     fscanf(file,"%d",&size);
        }

 //	printf(" computation_time= %e  total_time=%e \n",computation_time,0);
printf(" computation_time= %e  %d GCUPs=%lf\n",computation_time,total_size,( total_size)/computation_time/1000000000);



        return 0;
}


#undef STOP
#undef UP
#undef LEFT
#undef CROSSWISE
#undef DIRECTION
#undef CONTIN_UP
#undef CONTIN_LEFT
#undef ELEMENT



